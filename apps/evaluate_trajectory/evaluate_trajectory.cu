#include "hip/hip_runtime.h"
#include <chrono>
#include <atomic>
#include <iostream>

#include "util/arguments.h"
#include "mve/mesh_io_ply.h"
#include "mve/scene.h"
#include "acc/bvh_tree.h"
#include "cacc/point_cloud.h"
#include "cacc/util.h"
#include "cacc/matrix.h"
#include "cacc/bvh_tree.h"
#include "cacc/tracing.h"

#include "col/mpl_viridis.h"

#include "kernel.h"

typedef unsigned char uchar;

inline
uint divup(uint a, uint b) {
    return a / b  + (a % b != 0);
}

cacc::PointCloud<cacc::HOST>::Ptr
load_point_cloud(std::string const & path)
{
    mve::TriangleMesh::Ptr mesh;
    try {
        mesh = mve::geom::load_ply_mesh(path);
    } catch (std::exception& e) {
        std::cerr << "\tCould not load mesh: "<< e.what() << std::endl;
        std::exit(EXIT_FAILURE);
    }
    mesh->ensure_normals(true, true);

    std::vector<math::Vec3f> const & vertices = mesh->get_vertices();
    std::vector<math::Vec3f> const & normals = mesh->get_vertex_normals();

    cacc::PointCloud<cacc::HOST>::Ptr ret;
    ret = cacc::PointCloud<cacc::HOST>::create(vertices.size());
    cacc::PointCloud<cacc::HOST>::Data data = ret->cdata();
    for (std::size_t i = 0; i < vertices.size(); ++i) {
        data.vertices_ptr[i] = cacc::Vec3f(vertices[i].begin());
        data.normals_ptr[i] = cacc::Vec3f(normals[i].begin());
    }

    return ret;
}

acc::BVHTree<uint, math::Vec3f>::Ptr
load_mesh_as_bvh_tree(std::string const & path)
{
    mve::TriangleMesh::Ptr mesh;
    try {
        mesh = mve::geom::load_ply_mesh(path);
    } catch (std::exception& e) {
        std::cerr << "\tCould not load mesh: "<< e.what() << std::endl;
        std::exit(EXIT_FAILURE);
    }

    std::vector<math::Vec3f> const & vertices = mesh->get_vertices();
    std::vector<uint> const & faces = mesh->get_faces();
    return acc::BVHTree<uint, math::Vec3f>::create(faces, vertices);
}

void cpu(math::Matrix4f w2c, math::Matrix3f calib, math::Vec3f view_pos, int width, int height,
    acc::BVHTree<uint, math::Vec3f>::Ptr bvh_tree, cacc::PointCloud<cacc::HOST>::Ptr cloud,
    cacc::VectorArray<cacc::HOST, cacc::Vec2f>::Data dir_hist)
{
    std::chrono::time_point<std::chrono::high_resolution_clock> start, end;
    start = std::chrono::high_resolution_clock::now();

    #pragma omp for
    for (std::size_t i = 0; i < cloud->cdata().num_vertices; ++i) {
        cacc::Vec3f const & cv = cloud->cdata().vertices_ptr[i];
        math::Vec3f v; //TODO fix this mess
        for (int j = 0; j < 3; ++j) v[j] = cv[j];

        math::Vec3f v2c = view_pos - v;
        float n = v2c.norm();
        if (n > 45.0f) continue;
        math::Vec3f pt = calib * w2c.mult(v, 1.0f);
        math::Vec2f p(pt[0] / pt[2] - 0.5f, pt[1] / pt[2] - 0.5f);

        if (p[0] < 0.0f || width <= p[0] || p[1] < 0.0f || height <= p[1]) continue;

        acc::Ray<math::Vec3f> ray;
        ray.origin = v + v2c * 0.01f;
        ray.dir = v2c / n;
        ray.tmin = 0.0f;
        ray.tmax = inf;

        if (bvh_tree->intersect(ray)) continue;

        uint row = dir_hist.num_rows_ptr[i] + 1;

        if (row > dir_hist.max_rows) return;

        dir_hist.num_rows_ptr[i] = row;
        cacc::Vec2f dir(atan2(ray.dir[1], ray.dir[0]), acos(ray.dir[2]));
        dir_hist.data_ptr[row * dir_hist.pitch + i] = dir;
    }
    end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> diff = end - start;
    std::cout << "CPU: " << diff.count() << std::endl;
}

void gpu(math::Matrix4f w2c, math::Matrix3f calib, math::Vec3f view_pos, int width, int height,
    cacc::BVHTree<cacc::DEVICE>::Ptr dbvh_tree, cacc::PointCloud<cacc::DEVICE>::Ptr dcloud,
    cacc::VectorArray<cacc::DEVICE, cacc::Vec2f>::Ptr ddir_hist)
{
    std::chrono::time_point<std::chrono::high_resolution_clock> start, end;

    dim3 grid(divup(dcloud->cdata().num_vertices, KERNEL_BLOCK_SIZE));
    dim3 block(KERNEL_BLOCK_SIZE);

    start = std::chrono::high_resolution_clock::now();
    kernel<<<grid, block>>>(cacc::Mat4f(w2c.begin()), cacc::Mat3f(calib.begin()),
        cacc::Vec3f(view_pos.begin()), width, height,
        dbvh_tree->cdata(), dcloud->cdata(), ddir_hist->cdata());
    CHECK(hipDeviceSynchronize());
    end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> diff = end - start;
    std::cout << "GPU: " << diff.count() << std::endl;
}

void load_scene_as_trajectory(std::string const & path, std::vector<mve::CameraInfo> * trajectory) {
    mve::Scene::Ptr scene;
    try {
        scene = mve::Scene::create(path);
    } catch (std::exception& e) {
        std::cerr << "Could not open scene: " << e.what() << std::endl;
        std::exit(EXIT_FAILURE);
    }

    for (mve::View::Ptr const & view : scene->get_views()) {
        if (view == nullptr) continue;
        trajectory->push_back(view->get_camera());
    }
}

struct Arguments {
    std::string scene;
    std::string proxy_mesh;
    std::string proxy_cloud;
};

Arguments parse_args(int argc, char **argv) {
    util::Arguments args;
    args.set_exit_on_error(true);
    args.set_nonopt_maxnum(3);
    args.set_nonopt_minnum(3);
    args.set_usage("Usage: " + std::string(argv[0]) + " [OPTS] SCENE PROXY_MESH PROXY_CLOUD");
    args.set_description("Evaluate trajectory");
    args.parse(argc, argv);

    Arguments conf;
    conf.scene = args.get_nth_nonopt(0);
    conf.proxy_mesh = args.get_nth_nonopt(1);
    conf.proxy_cloud = args.get_nth_nonopt(2);

    for (util::ArgResult const* i = args.next_option();
         i != nullptr; i = args.next_option()) {
        switch (i->opt->sopt) {
        default:
            throw std::invalid_argument("Invalid option");
        }
    }

    return conf;
}

int main(int argc, char * argv[])
{
    Arguments args = parse_args(argc, argv);

    cacc::select_cuda_device(3, 5);

    acc::BVHTree<uint, math::Vec3f>::Ptr bvh_tree;
    bvh_tree = load_mesh_as_bvh_tree(args.proxy_mesh);
    cacc::BVHTree<cacc::DEVICE>::Ptr dbvh_tree;
    dbvh_tree = cacc::BVHTree<cacc::DEVICE>::create<uint, math::Vec3f>(bvh_tree);
    cacc::tracing::load_textures(dbvh_tree->cdata());

    cacc::PointCloud<cacc::HOST>::Ptr cloud;
    cloud = load_point_cloud(args.proxy_cloud);
    cacc::PointCloud<cacc::DEVICE>::Ptr dcloud;
    dcloud = cacc::PointCloud<cacc::DEVICE>::create<cacc::HOST>(cloud);

    uint num_vertices = dcloud->cdata().num_vertices;
    uint max_cameras = 20;

    cacc::VectorArray<cacc::HOST, cacc::Vec2f>::Ptr dir_hist;
    dir_hist = cacc::VectorArray<cacc::HOST, cacc::Vec2f>::create(num_vertices, max_cameras);
    cacc::VectorArray<cacc::DEVICE, cacc::Vec2f>::Ptr ddir_hist;
    ddir_hist = cacc::VectorArray<cacc::DEVICE, cacc::Vec2f>::create(num_vertices, max_cameras);

    std::vector<mve::CameraInfo> trajectory;
    load_scene_as_trajectory(args.scene, &trajectory);

    int width = 1920;
    int height = 1080;
    math::Matrix4f w2c;
    math::Matrix3f calib;
    math::Vec3f view_pos(0.0f);

    for (mve::CameraInfo const & cam : trajectory) {
        cam.fill_calibration(calib.begin(), width, height);
        cam.fill_cam_to_world(w2c.begin());
        cam.fill_camera_pos(view_pos.begin());

        cpu(w2c, calib, view_pos, width, height, bvh_tree, cloud, dir_hist->cdata());
        gpu(w2c, calib, view_pos, width, height, dbvh_tree, dcloud, ddir_hist);
    }


    mve::TriangleMesh::Ptr mesh;
    try {
        mesh = mve::geom::load_ply_mesh(args.proxy_cloud);
    } catch (std::exception& e) {
        std::cerr << "\tCould not load mesh: "<< e.what() << std::endl;
        std::exit(EXIT_FAILURE);
    }

    std::vector<math::Vec4f> colors = mesh->get_vertex_colors();
    colors.resize(num_vertices);
    for (std::size_t i = 0; i < num_vertices; ++i) {
        uint samples = dir_hist->cdata().num_rows_ptr[i];
        uchar quality = samples / max_cameras;
        math::Vec3f color(col::maps::viridis[quality]);
        colors[i] = math::Vec4f(color[0], color[1], color[2], 1.0f);
    }

    mve::geom::SavePLYOptions opts;
    opts.write_vertex_colors = true;
    mve::geom::save_ply_mesh(mesh, "/tmp/test.ply", opts);
}
