#include "hip/hip_runtime.h"
#include <chrono>
#include <atomic>
#include <iostream>

#include <hip/hip_runtime.h>

#include "util/arguments.h"

#include "util/io.h"

#include "mve/mesh_io_ply.h"
#include "mve/scene.h"

#include "acc/bvh_tree.h"

#include "cacc/point_cloud.h"
#include "cacc/util.h"
#include "cacc/math.h"
#include "cacc/matrix.h"
#include "cacc/tracing.h"
#include "cacc/reduction.h"

#include "col/mpl_viridis.h"

#include "eval/kernels.h"

typedef unsigned char uchar;

struct Arguments {
    std::string scene;
    std::string proxy_mesh;
    std::string proxy_cloud;
    std::string export_cloud;
    float max_distance;
};

Arguments parse_args(int argc, char **argv) {
    util::Arguments args;
    args.set_exit_on_error(true);
    args.set_nonopt_maxnum(3);
    args.set_nonopt_minnum(3);
    args.set_usage("Usage: " + std::string(argv[0]) + " [OPTS] SCENE PROXY_MESH PROXY_CLOUD");
    args.add_option('e', "export", true, "export per surface point reconstructability as point cloud");
    args.add_option('\0', "max-distance", true, "maximum distance to surface [80.0]");
    args.set_description("Evaluate trajectory");
    args.parse(argc, argv);

    Arguments conf;
    conf.scene = args.get_nth_nonopt(0);
    conf.proxy_mesh = args.get_nth_nonopt(1);
    conf.proxy_cloud = args.get_nth_nonopt(2);
    conf.max_distance = 2.0f;

    for (util::ArgResult const* i = args.next_option();
         i != nullptr; i = args.next_option()) {
        switch (i->opt->sopt) {
        case 'e':
            conf.export_cloud = i->arg;
        break;
        case '\0':
            if (i->opt->lopt == "max-distance") {
                conf.max_distance = i->get_arg<float>();
            } else {
                throw std::invalid_argument("Invalid option");
            }
        break;
        default:
            throw std::invalid_argument("Invalid option");
        }
    }

    return conf;
}

int main(int argc, char * argv[])
{
    Arguments args = parse_args(argc, argv);

    cacc::select_cuda_device(3, 5);

    cacc::BVHTree<cacc::DEVICE>::Ptr dbvh_tree;
    {
        acc::BVHTree<uint, math::Vec3f>::Ptr bvh_tree;
        bvh_tree = load_mesh_as_bvh_tree(args.proxy_mesh);
        dbvh_tree = cacc::BVHTree<cacc::DEVICE>::create<uint, math::Vec3f>(bvh_tree);
    }
    cacc::tracing::bind_textures(dbvh_tree->cdata());

    cacc::PointCloud<cacc::HOST>::Ptr cloud;
    cloud = load_point_cloud(args.proxy_cloud);
    cacc::PointCloud<cacc::DEVICE>::Ptr dcloud;
    dcloud = cacc::PointCloud<cacc::DEVICE>::create<cacc::HOST>(cloud);

    uint num_verts = dcloud->cdata().num_vertices;
    uint max_cameras = 20;

    cacc::VectorArray<cacc::Vec3f, cacc::DEVICE>::Ptr ddir_hist;
    ddir_hist = cacc::VectorArray<cacc::Vec3f, cacc::DEVICE>::create(num_verts, max_cameras);
    cacc::Array<float, cacc::DEVICE>::Ptr drecons;
    drecons = cacc::Array<float, cacc::DEVICE>::create(num_verts);
    drecons->null();

    std::vector<mve::CameraInfo> trajectory;
    load_scene_as_trajectory(args.scene, &trajectory);

    int width = 1920;
    int height = 1080;
    math::Matrix4f w2c;
    math::Matrix3f calib;
    math::Vec3f view_pos(0.0f);

    std::chrono::time_point<std::chrono::high_resolution_clock> start, end;

    start = std::chrono::high_resolution_clock::now();
    {
        hipStream_t stream;
        hipStreamCreate(&stream);
        dim3 grid(cacc::divup(num_verts, KERNEL_BLOCK_SIZE));
        dim3 block(KERNEL_BLOCK_SIZE);

        for (mve::CameraInfo const & cam : trajectory) {
            cam.fill_calibration(calib.begin(), width, height);
            cam.fill_world_to_cam(w2c.begin());
            cam.fill_camera_pos(view_pos.begin());

            populate_direction_histogram<<<grid, block, 0, stream>>>(
                cacc::Vec3f(view_pos.begin()), args.max_distance,
                cacc::Mat4f(w2c.begin()), cacc::Mat3f(calib.begin()), width, height,
                dbvh_tree->cdata(), dcloud->cdata(), drecons->cdata(), ddir_hist->cdata()
            );
        }

        hipStreamDestroy(stream);
        CHECK(hipDeviceSynchronize());
    }
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff = end - start;
    std::cout << "GPU: " << diff.count() << std::endl;

#if 0
    {
        dim3 grid(cacc::divup(num_verts, KERNEL_BLOCK_SIZE));
        dim3 block(KERNEL_BLOCK_SIZE);
        evaluate_histogram<<<grid, block>>>(ddir_hist->cdata());
        CHECK(hipDeviceSynchronize());
    }
#endif

    if (!args.export_cloud.empty()) {
        mve::TriangleMesh::Ptr mesh;
        try {
            mesh = mve::geom::load_ply_mesh(args.proxy_cloud);
        } catch (std::exception& e) {
            std::cerr << "\tCould not load mesh: "<< e.what() << std::endl;
            std::exit(EXIT_FAILURE);
        }

        std::vector<float> & values = mesh->get_vertex_values();
        values.resize(num_verts);

        std::cout << "GPU: " << cacc::sum(drecons) / num_verts << std::endl;

        cacc::Array<float, cacc::HOST> recons(*drecons);
        cacc::Array<float, cacc::HOST>::Data const & data = recons.cdata();
        for (std::size_t i = 0; i < num_verts; ++i) {
            values[i] = data.data_ptr[i];
        }

        float sum = std::accumulate(values.begin(), values.end(), 1.0f);
        std::cout << "CPU: " << sum / num_verts << std::endl;

        mve::geom::SavePLYOptions opts;
        opts.write_vertex_values = true;
        mve::geom::save_ply_mesh(mesh, args.export_cloud, opts);
    }
}
