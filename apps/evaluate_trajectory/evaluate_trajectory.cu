#include <chrono>
#include <atomic>
#include <iostream>

#include <hip/hip_runtime.h>

#include "util/arguments.h"

#include "util/io.h"

#include "mve/mesh_io_ply.h"
#include "mve/scene.h"

#include "acc/bvh_tree.h"

#include "cacc/point_cloud.h"
#include "cacc/util.h"
#include "cacc/math.h"
#include "cacc/matrix.h"
#include "cacc/tracing.h"

#include "col/mpl_viridis.h"

#include "eval/kernels.h"

typedef unsigned char uchar;

struct Arguments {
    std::string scene;
    std::string proxy_mesh;
    std::string proxy_cloud;
    std::string export_cloud;
    float max_distance;
};

Arguments parse_args(int argc, char **argv) {
    util::Arguments args;
    args.set_exit_on_error(true);
    args.set_nonopt_maxnum(3);
    args.set_nonopt_minnum(3);
    args.set_usage("Usage: " + std::string(argv[0]) + " [OPTS] SCENE PROXY_MESH PROXY_CLOUD");
    args.add_option('e', "export", true, "export per surface point reconstructability as point cloud");
    args.set_description("Evaluate trajectory");
    args.parse(argc, argv);

    Arguments conf;
    conf.scene = args.get_nth_nonopt(0);
    conf.proxy_mesh = args.get_nth_nonopt(1);
    conf.proxy_cloud = args.get_nth_nonopt(2);
    conf.max_distance = 2.0f;

    for (util::ArgResult const* i = args.next_option();
         i != nullptr; i = args.next_option()) {
        switch (i->opt->sopt) {
        case 'e':
            conf.export_cloud = i->arg;
        break;
        default:
            throw std::invalid_argument("Invalid option");
        }
    }

    return conf;
}

int main(int argc, char * argv[])
{
    Arguments args = parse_args(argc, argv);

    cacc::select_cuda_device(3, 5);

    cacc::BVHTree<cacc::DEVICE>::Ptr dbvh_tree;
    {
        acc::BVHTree<uint, math::Vec3f>::Ptr bvh_tree;
        bvh_tree = load_mesh_as_bvh_tree(args.proxy_mesh);
        dbvh_tree = cacc::BVHTree<cacc::DEVICE>::create<uint, math::Vec3f>(bvh_tree);
    }
    cacc::tracing::bind_textures(dbvh_tree->cdata());

    cacc::PointCloud<cacc::HOST>::Ptr cloud;
    cloud = load_point_cloud(args.proxy_cloud);
    cacc::PointCloud<cacc::DEVICE>::Ptr dcloud;
    dcloud = cacc::PointCloud<cacc::DEVICE>::create<cacc::HOST>(cloud);

    uint num_vertices = dcloud->cdata().num_vertices;
    uint max_cameras = 20;

    cacc::VectorArray<cacc::Vec3f, cacc::HOST>::Ptr hdir_hist;
    hdir_hist = cacc::VectorArray<cacc::Vec3f, cacc::HOST>::create(num_vertices, max_cameras);
    cacc::VectorArray<cacc::Vec3f, cacc::DEVICE>::Ptr ddir_hist;
    ddir_hist = cacc::VectorArray<cacc::Vec3f, cacc::DEVICE>::create(num_vertices, max_cameras);

    std::vector<mve::CameraInfo> trajectory;
    load_scene_as_trajectory(args.scene, &trajectory);

    int width = 1920;
    int height = 1080;
    math::Matrix4f w2c;
    math::Matrix3f calib;
    math::Vec3f view_pos(0.0f);

    std::chrono::time_point<std::chrono::high_resolution_clock> start, end;

    start = std::chrono::high_resolution_clock::now();
    {
        hipStream_t stream;
        hipStreamCreate(&stream);
        dim3 grid(cacc::divup(num_vertices, KERNEL_BLOCK_SIZE));
        dim3 block(KERNEL_BLOCK_SIZE);

        for (mve::CameraInfo const & cam : trajectory) {
            cam.fill_calibration(calib.begin(), width, height);
            cam.fill_world_to_cam(w2c.begin());
            cam.fill_camera_pos(view_pos.begin());

            populate_histogram<<<grid, block, 0, stream>>>(
                cacc::Vec3f(view_pos.begin()), args.max_distance,
                cacc::Mat4f(w2c.begin()), cacc::Mat3f(calib.begin()), width, height,
                dbvh_tree->cdata(), dcloud->cdata(), ddir_hist->cdata()
            );
        }
        CHECK(hipDeviceSynchronize());
    }
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff = end - start;
    std::cout << "GPU: " << diff.count() << std::endl;

    {
        dim3 grid(cacc::divup(num_vertices, KERNEL_BLOCK_SIZE));
        dim3 block(KERNEL_BLOCK_SIZE);
        evaluate_histogram<<<grid, block>>>(ddir_hist->cdata());
        CHECK(hipDeviceSynchronize());
    }

    *hdir_hist = *ddir_hist;

    if (!args.export_cloud.empty()) {
        mve::TriangleMesh::Ptr mesh;
        try {
            mesh = mve::geom::load_ply_mesh(args.proxy_cloud);
        } catch (std::exception& e) {
            std::cerr << "\tCould not load mesh: "<< e.what() << std::endl;
            std::exit(EXIT_FAILURE);
        }

        std::vector<float> & values = mesh->get_vertex_values();
        values.resize(num_vertices);

        cacc::VectorArray<cacc::Vec3f, cacc::HOST>::Data const & dir_hist = hdir_hist->cdata();
        int const stride = dir_hist.pitch / sizeof(cacc::Vec3f);
        #pragma omp parallel for
        for (std::size_t i = 0; i < num_vertices; ++i) {
            values[i] = dir_hist.data_ptr[(max_cameras - 1) * stride + i][3];
        }
        mve::geom::SavePLYOptions opts;
        opts.write_vertex_values = true;
        mve::geom::save_ply_mesh(mesh, args.export_cloud, opts);
    }
}
