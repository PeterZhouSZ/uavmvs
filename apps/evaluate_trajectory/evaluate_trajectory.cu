#include "hip/hip_runtime.h"
#include <chrono>
#include <atomic>
#include <iostream>

#include <hip/hip_runtime.h>

#include "util/arguments.h"
#include "util/file_system.h"

#include "util/io.h"

#include "mve/mesh_io_ply.h"
#include "mve/scene.h"

#include "acc/bvh_tree.h"

#include "cacc/point_cloud.h"
#include "cacc/util.h"
#include "cacc/math.h"
#include "cacc/matrix.h"
#include "cacc/reduction.h"

#include "col/mpl_viridis.h"

#include "eval/kernels.h"

#include "utp/trajectory.h"
#include "utp/trajectory_io.h"

typedef unsigned char uchar;

struct Arguments {
    std::string trajectory;
    std::string proxy_mesh;
    std::string proxy_cloud;
    std::string export_cloud;
    float max_distance;
};

Arguments parse_args(int argc, char **argv) {
    util::Arguments args;
    args.set_exit_on_error(true);
    args.set_nonopt_maxnum(3);
    args.set_nonopt_minnum(3);
    args.set_usage("Usage: " + std::string(argv[0]) + " [OPTS] TRAJECTORY/SCENE PROXY_MESH PROXY_CLOUD");
    args.add_option('e', "export", true, "export per surface point reconstructability as point cloud");
    args.add_option('\0', "max-distance", true, "maximum distance to surface [80.0]");
    args.set_description("Evaluate trajectory");
    args.parse(argc, argv);

    Arguments conf;
    conf.trajectory = args.get_nth_nonopt(0);
    conf.proxy_mesh = args.get_nth_nonopt(1);
    conf.proxy_cloud = args.get_nth_nonopt(2);
    conf.max_distance = 80.0f;

    for (util::ArgResult const* i = args.next_option();
         i != nullptr; i = args.next_option()) {
        switch (i->opt->sopt) {
        case 'e':
            conf.export_cloud = i->arg;
        break;
        case '\0':
            if (i->opt->lopt == "max-distance") {
                conf.max_distance = i->get_arg<float>();
            } else {
                throw std::invalid_argument("Invalid option");
            }
        break;
        default:
            throw std::invalid_argument("Invalid option");
        }
    }

    return conf;
}

int main(int argc, char * argv[])
{
    Arguments args = parse_args(argc, argv);

    cacc::select_cuda_device(3, 5);

    std::vector<mve::CameraInfo> trajectory;
    if (util::fs::dir_exists(args.trajectory.c_str())) {
        load_scene_as_trajectory(args.trajectory, &trajectory);
    } else if (util::fs::file_exists(args.trajectory.c_str())) {
        utp::load_trajectory(args.trajectory, &trajectory);
    } else {
        std::cerr << "Could not load trajectory" << std::endl;
        return EXIT_FAILURE;
    }

    cacc::BVHTree<cacc::DEVICE>::Ptr dbvh_tree;
    {
        acc::BVHTree<uint, math::Vec3f>::Ptr bvh_tree;
        bvh_tree = load_mesh_as_bvh_tree(args.proxy_mesh);
        dbvh_tree = cacc::BVHTree<cacc::DEVICE>::create<uint, math::Vec3f>(bvh_tree);
    }

    cacc::PointCloud<cacc::HOST>::Ptr cloud;
    cloud = load_point_cloud(args.proxy_cloud);
    cacc::PointCloud<cacc::DEVICE>::Ptr dcloud;
    dcloud = cacc::PointCloud<cacc::DEVICE>::create<cacc::HOST>(cloud);

    uint num_verts = dcloud->cdata().num_vertices;
    uint max_cameras = 20;

    cacc::VectorArray<cacc::Vec3f, cacc::DEVICE>::Ptr ddir_hist;
    ddir_hist = cacc::VectorArray<cacc::Vec3f, cacc::DEVICE>::create(num_verts, max_cameras);
    cacc::Array<float, cacc::DEVICE>::Ptr drecons;
    drecons = cacc::Array<float, cacc::DEVICE>::create(num_verts);
    drecons->null();

    std::cout << '\n';

    int width = 1920;
    int height = 1080;
    math::Matrix4f w2c;
    math::Matrix3f calib;
    math::Vec3f view_pos(0.0f);

    std::chrono::time_point<std::chrono::high_resolution_clock> start, end;

    std::cout << "Computing reconstuctability" << std::endl;
    start = std::chrono::high_resolution_clock::now();
    {
        hipStream_t stream;
        hipStreamCreate(&stream);
        dim3 grid(cacc::divup(num_verts, KERNEL_BLOCK_SIZE));
        dim3 block(KERNEL_BLOCK_SIZE);

        for (mve::CameraInfo const & cam : trajectory) {
            cam.fill_calibration(calib.begin(), width, height);
            cam.fill_world_to_cam(w2c.begin());
            cam.fill_camera_pos(view_pos.begin());

            update_direction_histogram<<<grid, block, 0, stream>>>(
                true, cacc::Vec3f(view_pos.begin()), args.max_distance,
                cacc::Mat4f(w2c.begin()), cacc::Mat3f(calib.begin()), width, height,
                dbvh_tree->accessor(), dcloud->cdata(), ddir_hist->cdata()
            );
        }

        hipStreamDestroy(stream);
        CHECK(hipDeviceSynchronize());
    }
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff = end - start;
    std::cout << "  GPU: " << diff.count() << 's' << std::endl;

    {
        dim3 grid(cacc::divup(num_verts, 2));
        dim3 block(32, 2);
        process_direction_histogram<<<grid, block>>>(
            ddir_hist->cdata());
    }

    {
        dim3 grid(cacc::divup(num_verts, KERNEL_BLOCK_SIZE));
        dim3 block(KERNEL_BLOCK_SIZE);
        evaluate_direction_histogram<<<grid, block>>>(ddir_hist->cdata(), drecons->cdata());
        CHECK(hipDeviceSynchronize());
    }

    std::vector<float> values(num_verts);

    cacc::Array<float, cacc::HOST> recons(*drecons);
    cacc::Array<float, cacc::HOST>::Data const & data = recons.cdata();
    for (std::size_t i = 0; i < num_verts; ++i) {
        values[i] = data.data_ptr[i];
    }

    std::cout << "Average reconstructability" << std::endl;
    std::cout << "  GPU:\n"
        << "  " << cacc::reduction::sum(drecons) / num_verts << '\n'
        << "  " << cacc::reduction::min(drecons) << '\n'
        << "  " << cacc::reduction::max(drecons) << '\n'
        << std::endl;
    std::cout << "  CPU:\n"
        << "  " << std::accumulate(values.begin(), values.end(), 1.0f) / num_verts << '\n'
        << "  " << *std::min_element(values.begin(), values.end()) << '\n'
        << "  " << *std::max_element(values.begin(), values.end()) << '\n'
        << std::endl;

    std::cout << "Length: " << utp::length(trajectory) << '\n' << std::endl;

    if (!args.export_cloud.empty()) {
        mve::TriangleMesh::Ptr mesh;
        try {
            mesh = mve::geom::load_ply_mesh(args.proxy_cloud);
        } catch (std::exception& e) {
            std::cerr << "\tCould not load mesh: "<< e.what() << std::endl;
            std::exit(EXIT_FAILURE);
        }

        std::vector<float> & ovalues = mesh->get_vertex_values();
        ovalues.assign(values.begin(), values.end());

        mve::geom::SavePLYOptions opts;
        opts.write_vertex_values = true;
        mve::geom::save_ply_mesh(mesh, args.export_cloud, opts);
    }
}
