#include <fstream>
#include <iostream>

#include "fmt/format.h"

#include "util/system.h"
#include "util/arguments.h"
#include "util/choices.h"

#include "mve/camera.h"
#include "mve/mesh_io_ply.h"
#include "mve/image_io.h"

#include "cacc/util.h"
#include "cacc/math.h"
#include "cacc/tracing.h"
#include "cacc/nnsearch.h"

#include "eval/kernels.h"

struct Arguments {
    std::string proxy_mesh;
    std::string proxy_cloud;
    std::string proxy_sphere;
    std::string trajectory;
    float min_distance;
    float max_velocity;
};

Arguments parse_args(int argc, char **argv) {
    util::Arguments args;
    args.set_exit_on_error(true);
    args.set_nonopt_minnum(4);
    args.set_nonopt_maxnum(4);
    args.set_usage("Usage: " + std::string(argv[0])
        + " [OPTS] PROXY_MESH PROXY_CLOUD PROXY_SPHERE TRAJECTORY");
    args.set_description("Plans a trajectory maximizing reconstructability");
    args.parse(argc, argv);

    Arguments conf;
    conf.proxy_mesh = args.get_nth_nonopt(0);
    conf.proxy_cloud = args.get_nth_nonopt(1);
    conf.proxy_sphere = args.get_nth_nonopt(2);
    conf.trajectory = args.get_nth_nonopt(3);

    for (util::ArgResult const* i = args.next_option();
         i != 0; i = args.next_option()) {
        switch (i->opt->sopt) {
        default:
            throw std::invalid_argument("Invalid option");
        }
    }

    return conf;
}

void load_trajectory(std::string const & path,
        std::vector<mve::CameraInfo> * trajectory)
{
    std::ifstream in(path.c_str());
    if (!in.good()) throw std::runtime_error("Could not open trajectory file");
    std::size_t length;
    in >> length;

    trajectory->resize(length);

    for (std::size_t i = 0; i < length; ++i) {
        math::Vec3f pos;
        for (int j = 0; j < 3; ++j) {
            in >> pos[j];
        }
        math::Matrix3f rot;
        for (int j = 0; j < 9; ++j) {
            in >> rot[j];
        }
        math::Vec3f trans = -rot * pos;

        mve::CameraInfo & cam = trajectory->at(i);
        cam.flen = 0.86f; //TODO save and read from file
        std::copy(trans.begin(), trans.end(), cam.trans);
        std::copy(rot.begin(), rot.end(), cam.rot);
    }

    if (in.fail()) {
        in.close();
        throw std::runtime_error("Invalid trajectory file");
    }

    in.close();
}


acc::KDTree<3, uint>::Ptr
load_mesh_as_kd_tree(std::string const & path)
{
    mve::TriangleMesh::Ptr mesh;
    try {
        mesh = mve::geom::load_ply_mesh(path);
    } catch (std::exception& e) {
        std::cerr << "\tCould not load mesh: "<< e.what() << std::endl;
        std::exit(EXIT_FAILURE);
    }

    std::vector<math::Vec3f> const & vertices = mesh->get_vertices();
    return acc::KDTree<3, uint>::create(vertices);
}

acc::BVHTree<uint, math::Vec3f>::Ptr
load_mesh_as_bvh_tree(std::string const & path)
{
    mve::TriangleMesh::Ptr mesh;
    try {
        mesh = mve::geom::load_ply_mesh(path);
    } catch (std::exception& e) {
        std::cerr << "\tCould not load mesh: "<< e.what() << std::endl;
        std::exit(EXIT_FAILURE);
    }

    std::vector<math::Vec3f> const & vertices = mesh->get_vertices();
    std::vector<uint> const & faces = mesh->get_faces();
    return acc::BVHTree<uint, math::Vec3f>::create(faces, vertices);
}

cacc::PointCloud<cacc::HOST>::Ptr
load_point_cloud(std::string const & path)
{
    mve::TriangleMesh::Ptr mesh;
    try {
        mesh = mve::geom::load_ply_mesh(path);
    } catch (std::exception& e) {
        std::cerr << "\tCould not load mesh: " << e.what() << std::endl;
        std::exit(EXIT_FAILURE);
    }
    mesh->ensure_normals(true, true);

    std::vector<math::Vec3f> const & vertices = mesh->get_vertices();
    std::vector<math::Vec3f> const & normals = mesh->get_vertex_normals();

    cacc::PointCloud<cacc::HOST>::Ptr ret;
    ret = cacc::PointCloud<cacc::HOST>::create(vertices.size());
    cacc::PointCloud<cacc::HOST>::Data data = ret->cdata();
    for (std::size_t i = 0; i < vertices.size(); ++i) {
        data.vertices_ptr[i] = cacc::Vec3f(vertices[i].begin());
        data.normals_ptr[i] = cacc::Vec3f(normals[i].begin());
    }

    return ret;
}

int main(int argc, char **argv) {
    util::system::register_segfault_handler();
    util::system::print_build_timestamp(argv[0]);

    Arguments args = parse_args(argc, argv);

    cacc::select_cuda_device(3, 5);

    mve::TriangleMesh::Ptr mesh;
    try {
        mesh = mve::geom::load_ply_mesh(args.proxy_sphere);
    } catch (std::exception& e) {
        std::cerr << "\tCould not load mesh: "<< e.what() << std::endl;
        std::exit(EXIT_FAILURE);
    }

    std::vector<math::Vec3f> & overtices = mesh->get_vertices();
    std::vector<math::Vec3f> vertices = mesh->get_vertices();
    std::vector<float> & ovalues = mesh->get_vertex_values();
    ovalues.resize(vertices.size());

    acc::KDTree<3u, uint>::Ptr kd_tree;
    kd_tree = load_mesh_as_kd_tree(args.proxy_sphere);
    cacc::KDTree<3u, cacc::DEVICE>::Ptr dkd_tree;
    dkd_tree = cacc::KDTree<3u, cacc::DEVICE>::create<uint>(kd_tree);
    cacc::nnsearch::bind_textures(dkd_tree->cdata());

    acc::BVHTree<uint, math::Vec3f>::Ptr bvh_tree;
    bvh_tree = load_mesh_as_bvh_tree(args.proxy_mesh);
    cacc::BVHTree<cacc::DEVICE>::Ptr dbvh_tree;
    dbvh_tree = cacc::BVHTree<cacc::DEVICE>::create<uint, math::Vec3f>(bvh_tree);
    cacc::tracing::bind_textures(dbvh_tree->cdata());

    cacc::PointCloud<cacc::HOST>::Ptr cloud;
    cloud = load_point_cloud(args.proxy_cloud);
    cacc::PointCloud<cacc::DEVICE>::Ptr dcloud;
    dcloud = cacc::PointCloud<cacc::DEVICE>::create<cacc::HOST>(cloud);

    uint num_vertices = dcloud->cdata().num_vertices;
    uint max_cameras = 20;

    cacc::VectorArray<cacc::Vec2f, cacc::DEVICE>::Ptr ddir_hist;
    ddir_hist = cacc::VectorArray<cacc::Vec2f, cacc::DEVICE>::create(num_vertices, max_cameras);
    cacc::VectorArray<float, cacc::HOST>::Ptr con_hist;
    con_hist = cacc::VectorArray<float, cacc::HOST>::create(ovalues.size(), 2);
    cacc::VectorArray<float, cacc::HOST>::Data data = con_hist->cdata();
    cacc::VectorArray<float, cacc::DEVICE>::Ptr dcon_hist;
    dcon_hist = cacc::VectorArray<float, cacc::DEVICE>::create(ovalues.size(), 2);

    cacc::Image<float, cacc::DEVICE>::Ptr dhist;
    dhist = cacc::Image<float, cacc::DEVICE>::create(360, 180);

    math::Vec3f pos;
    math::Matrix3f calib;
    int width = 1920;
    int height = 1080;

    int cnt = 0;

    std::vector<mve::CameraInfo> trajectory;
    load_trajectory(args.trajectory, &trajectory);

    for (mve::CameraInfo const & cam : trajectory) {
        cam.fill_calibration(calib.begin(), width, height);
        cam.fill_camera_pos(pos.begin());

        //TODO write clear kernel
        for (std::size_t i = 0; i < ovalues.size(); ++i) {
            data.data_ptr[i] = 0.0f;
            data.data_ptr[i + data.pitch / sizeof(float)] = cacc::float_to_uint32(0.0f);
        }
        *dcon_hist = *con_hist;

        {
            dim3 grid(cacc::divup(num_vertices, KERNEL_BLOCK_SIZE));
            dim3 block(KERNEL_BLOCK_SIZE);
            populate_histogram<<<grid, block>>>(cacc::Vec3f(pos.begin()),
                dbvh_tree->cdata(), dcloud->cdata(), dkd_tree->cdata(),
                ddir_hist->cdata(), dcon_hist->cdata());
            CHECK(hipDeviceSynchronize());
        }

        {
            dim3 grid(cacc::divup(360, KERNEL_BLOCK_SIZE), 180);
            dim3 block(KERNEL_BLOCK_SIZE);
            evaluate_histogram<<<grid, block>>>(cacc::Mat3f(calib.begin()), width, height,
                dkd_tree->cdata(), dcon_hist->cdata(), dhist->cdata());
            CHECK(hipDeviceSynchronize());
        }

        {
            dim3 grid(cacc::divup(360, KERNEL_BLOCK_SIZE), 180);
            dim3 block(KERNEL_BLOCK_SIZE);
            evaluate_histogram<<<grid, block>>>(dkd_tree->cdata(),
               dhist->cdata(), dcon_hist->cdata());
            CHECK(hipDeviceSynchronize());
        }

        *con_hist = *dcon_hist;
        for (std::size_t i = 0; i < vertices.size(); ++i) {
            overtices[i] = vertices[i] + pos;
#if 1
            float * f = data.data_ptr + data.pitch / sizeof(float) + i;
            uint32_t v = reinterpret_cast<uint32_t&>(*f);
            ovalues[i] = cacc::uint32_to_float(v);
#else
            ovalues[i] = data.data_ptr[i];
#endif
        }

        mve::geom::SavePLYOptions opts;
        opts.write_vertex_values = true;
        std::string filename = fmt::format("/tmp/test-sphere-{:04d}.ply", cnt++);
        mve::geom::save_ply_mesh(mesh, filename, opts);
    }

    return EXIT_SUCCESS;
}
