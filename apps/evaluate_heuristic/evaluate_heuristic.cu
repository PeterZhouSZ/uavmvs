#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>

#include "util/system.h"
#include "util/arguments.h"

#include "util/io.h"
#include "util/numpy_io.h"

#include "cacc/util.h"
#include "cacc/math.h"

#include "eval/kernels.h"

#include "stat/correlations.h"

#include "mve/scene.h"
#include "mve/image.h"

struct Arguments {
    std::string scene;
    std::string image;
    std::string gt_mesh;
    std::string file;
    std::string recon_cloud;
    std::string obs_cloud;
    float max_distance;
    float target_recon;
};

typedef unsigned int uint;
typedef acc::BVHTree<uint, math::Vec3f> BVHTree;

Arguments parse_args(int argc, char **argv) {
    util::Arguments args;
    args.set_exit_on_error(true);
    args.set_nonopt_minnum(4);
    args.set_nonopt_maxnum(4);
    args.set_usage("Usage: " + std::string(argv[0]) +
        " [OPTS] SCENE IMAGE GT_MESH FILE");
    args.set_description("Evaluates Spearman's rank correlation between "
        "depth error and heuristic for multiple parameter sets.");
    args.add_option('r', "recon-cloud", true,
        "save cloud with predicted reconstructabilities");
    args.add_option('o', "obs-cloud", true,
        "save cloud with number of observations reconstructabilities");
    args.add_option('\0', "max-distance", true, "maximum distance to surface [80.0]");
    args.parse(argc, argv);

    Arguments conf;
    conf.scene = args.get_nth_nonopt(0);
    conf.image = args.get_nth_nonopt(1);
    conf.gt_mesh = args.get_nth_nonopt(2);
    conf.file = args.get_nth_nonopt(3);
    conf.max_distance = 80.0f;

    for (util::ArgResult const* i = args.next_option();
         i != 0; i = args.next_option()) {
        switch (i->opt->sopt) {
        case 'r':
            conf.recon_cloud = i->arg;
        break;
        case 'o':
            conf.obs_cloud = i->arg;
        break;
        case '\0':
            if (i->opt->lopt == "max-distance") {
                conf.max_distance = i->get_arg<float>();
            } else {
                throw std::invalid_argument("Invalid option");
            }
        break;
        default:
            throw std::invalid_argument("Invalid option");
        }
    }

    return conf;
}

template <int N> inline void
patch(mve::FloatImage::Ptr img, int x, int y, float (*ptr)[N][N]) {
    static_assert(N % 2 == 1, "Requires odd patch size");
    constexpr int e = N / 2;
    for (int i = -e; i <= e; ++i) {
        for (int j = -e; j <= e; ++j) {
            (*ptr)[e + j][e + i] = img->at(x + j, y + i, 0);
        }
    }
}

int main(int argc, char **argv) {
    util::system::register_segfault_handler();
    util::system::print_build_timestamp(argv[0]);

    Arguments args = parse_args(argc, argv);

    int device = cacc::select_cuda_device(3, 5);

    mve::Scene::Ptr scene;
    try {
        scene = mve::Scene::create(args.scene);
    } catch (std::exception& e) {
        std::cerr << "Could not open scene: " << e.what() << std::endl;
        std::exit(EXIT_FAILURE);
    }

    mve::TriangleMesh::Ptr mesh;
    try {
        mesh = mve::geom::load_ply_mesh(args.gt_mesh);
    } catch (std::exception& e) {
        std::cerr << "\tCould not load mesh: "<< e.what() << std::endl;
        std::exit(EXIT_FAILURE);
    }

    std::vector<math::Vec3f> const & vertices = mesh->get_vertices();
    std::vector<uint> const & faces = mesh->get_faces();
    std::cout << "Building BVH... " << std::flush;
    BVHTree::Ptr bvh_tree = BVHTree::create(faces, vertices);
    std::cout << "done." << std::endl;

    std::vector<float> errors;
    std::vector<math::Vec3f> verts;
    std::vector<math::Vec3f> normals;

    std::vector<mve::View::Ptr> views = scene->get_views();
    for (mve::View::Ptr & view : views) {
        if (view == nullptr) continue;
        if (!view->has_image(args.image, mve::IMAGE_TYPE_FLOAT)) {
            std::cerr << "Warning view " << view->get_name()
                << " has no image " << args.image << std::endl;
            continue;
        }

        mve::FloatImage::Ptr dmap = view->get_float_image(args.image);

        mve::CameraInfo const & camera = view->get_camera();
        math::Vec3f origin;
        camera.fill_camera_pos(origin.begin());
        math::Matrix3f invcalib;
        camera.fill_inverse_calibration(invcalib.begin(),
            dmap->width(), dmap->height());
        math::Matrix3f c2w_rot;
        camera.fill_cam_to_world_rot(c2w_rot.begin());

        /* Ignore border - issues with kernel approaches. */
        int border = 0.01f * max(dmap->width(), dmap->height());
        for (int y = border; y < dmap->height() - border; ++y) {
            for (int x = border; x < dmap->width() - border; ++x) {
                float depth = dmap->at(x, y, 0);

                BVHTree::Ray ray;
                ray.origin = origin;
                math::Vec3f v = invcalib *
                    math::Vec3f ((float)x + 0.5f, (float)y + 0.5f, 1.0f);
                ray.dir = c2w_rot.mult(v.normalized()).normalize();
                ray.tmin = 0.0f;
                ray.tmax = std::numeric_limits<float>::infinity();

                /* Ground truth depth? */
                BVHTree::Hit hit;
                if (!bvh_tree->intersect(ray, &hit)) continue;

                verts.push_back(origin + (hit.t * ray.dir));
                math::Vec3f v0 = vertices[faces[hit.idx * 3]];
                math::Vec3f v1 = vertices[faces[hit.idx * 3 + 1]];
                math::Vec3f v2 = vertices[faces[hit.idx * 3 + 2]];
                normals.push_back((v2 - v0).cross(v1 - v0).normalize());

                //float depths[25];
                //patch(dmap, x, y, (float (*)[5][5])&depths);
                //if (std::any_of(depths, depths + 25,
                //        [] (float d) { return d == 0.0f; })) {
                if (depth == 0) {
                    errors.push_back(-1.0f);
                } else {
                    errors.push_back(std::abs(depth - (hit.t * ray.dir).norm()));
                }
            }
        }
    }

    /* Construct cloud for heuristic evaluation on GPU. */
    cacc::PointCloud<cacc::HOST>::Ptr cloud;
    cloud = cacc::PointCloud<cacc::HOST>::create(verts.size());
    cacc::PointCloud<cacc::HOST>::Data data = cloud->cdata();
    for (std::size_t i = 0; i < verts.size(); ++i) {
        data.vertices_ptr[i] = cacc::Vec3f(verts[i].begin());
        data.normals_ptr[i] = cacc::Vec3f(normals[i].begin());
        data.values_ptr[i] = 0.0f;
        data.qualities_ptr[i] = 1.0f;
    }
    cacc::PointCloud<cacc::DEVICE>::Ptr dcloud;
    dcloud = cacc::PointCloud<cacc::DEVICE>::create<cacc::HOST>(cloud);

    cacc::BVHTree<cacc::DEVICE>::Ptr dbvh_tree;
    dbvh_tree = cacc::BVHTree<cacc::DEVICE>::create<uint, math::Vec3f>(bvh_tree);

    uint num_verts = verts.size();
    uint max_cameras = 32;

    cacc::VectorArray<cacc::Vec3f, cacc::DEVICE>::Ptr dobs_rays;
    dobs_rays = cacc::VectorArray<cacc::Vec3f, cacc::DEVICE>::create(num_verts, max_cameras);
    cacc::Array<float, cacc::DEVICE>::Ptr drecons;
    drecons = cacc::Array<float, cacc::DEVICE>::create(num_verts);
    drecons->null();

    int width = 1920;
    int height = 1080;
    math::Matrix4f w2c;
    math::Matrix3f calib;
    math::Vec3f view_pos(0.0f);

    /* Populate view direction histograms. */
    {
        hipStream_t stream;
        hipStreamCreate(&stream);
        dim3 grid(cacc::divup(num_verts, KERNEL_BLOCK_SIZE));
        dim3 block(KERNEL_BLOCK_SIZE);

        for (mve::View::Ptr const & view : scene->get_views()) {
            if (view == nullptr) continue;

            mve::CameraInfo cam = view->get_camera();
            cam.fill_calibration(calib.begin(), width, height);
            cam.fill_world_to_cam(w2c.begin());
            cam.fill_camera_pos(view_pos.begin());

            update_observation_rays<<<grid, block, 0, stream>>>(
                true, cacc::Vec3f(view_pos.begin()), args.max_distance,
                cacc::Mat4f(w2c.begin()), cacc::Mat3f(calib.begin()), width, height,
                dbvh_tree->accessor(), dcloud->cdata(), dobs_rays->cdata()
            );
        }

        hipStreamDestroy(stream);
        CHECK(hipDeviceSynchronize());
    }

    {
        dim3 grid(cacc::divup(num_verts, 2));
        dim3 block(32, 2);
        process_observation_rays<<<grid, block>>>(
           dobs_rays->cdata());
    }

    std::vector<float> heuristics(verts.size());
    std::vector<float> observations(verts.size());

    float m_k = 8;
    float m_x0 =4;
    float t_k = 32;
    float t_x0 = 16;
    {
        configure_heuristic(m_k, m_x0, t_k, t_x0);
        CHECK(hipDeviceSynchronize());

        dim3 grid(cacc::divup(num_verts, KERNEL_BLOCK_SIZE));
        dim3 block(KERNEL_BLOCK_SIZE);
        evaluate_observation_rays<<<grid, block>>>(dobs_rays->cdata(),
            drecons->cdata());
        CHECK(hipDeviceSynchronize());

        {
            cacc::Array<float, cacc::HOST> recons(*drecons);
            cacc::Array<float, cacc::HOST>::Data const & data = recons.cdata();
            CHECK(hipDeviceSynchronize());

            for (std::size_t k = 0; k < data.num_values; ++k) {
                heuristics[k] = data.data_ptr[k];
            }
        }
        std::cout << stat::spearmans_rank_correlation(heuristics, errors) << std::endl;

        {
            cacc::VectorArray<cacc::Vec3f, cacc::HOST> obs_rays(*dobs_rays);
            cacc::VectorArray<cacc::Vec3f, cacc::HOST>::Data const & data = obs_rays.cdata();
            CHECK(hipDeviceSynchronize());

            for (std::size_t k = 0; k < data.num_cols; ++k) {
                observations[k] = data.num_rows_ptr[k];
            }
        }

        save_numpy_file(heuristics, errors, observations, args.file);
    }

    if (!args.recon_cloud.empty() || !args.obs_cloud.empty()) {
        mve::TriangleMesh::Ptr mesh = mve::TriangleMesh::create();

        mesh->get_vertices().assign(verts.begin(), verts.end());

        mve::geom::SavePLYOptions opts;
        opts.write_vertex_values = true;

        if (!args.recon_cloud.empty()) {
            mesh->get_vertex_values().assign(heuristics.begin(), heuristics.end());
            mve::geom::save_ply_mesh(mesh, args.recon_cloud, opts);
        }

        if (!args.obs_cloud.empty()) {
            mesh->get_vertex_values().assign(observations.begin(), observations.end());
            mve::geom::save_ply_mesh(mesh, args.obs_cloud, opts);
        }
    }

    return EXIT_SUCCESS;
}
