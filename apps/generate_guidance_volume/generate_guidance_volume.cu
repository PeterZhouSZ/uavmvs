#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>

#include "fmt/format.h"

#include "util/system.h"
#include "util/arguments.h"
#include "util/file_system.h"

#include "mve/camera.h"
#include "mve/mesh_io_ply.h"
#include "mve/image_io.h"
#include "mve/image_tools.h"

#include "acc/primitives.h"

#include "cacc/math.h"
#include "cacc/util.h"
#include "cacc/bvh_tree.h"
#include "cacc/tracing.h"
#include "cacc/nnsearch.h"
#include "cacc/point_cloud.h"

#include "util/io.h"
#include "util/progress_counter.h"
#include "util/itos.h"

#include "geom/sphere.h"

#include "eval/kernels.h"

constexpr float lowest = std::numeric_limits<float>::lowest();

struct Arguments {
    std::string proxy_mesh;
    std::string proxy_cloud;
    std::string airspace_mesh;
    std::string ovolume;
    float resolution;
    float max_distance;
    float max_altitude;
};

Arguments parse_args(int argc, char **argv) {
    util::Arguments args;
    args.set_exit_on_error(true);
    args.set_nonopt_minnum(4);
    args.set_nonopt_maxnum(4);
    args.set_usage("Usage: " + std::string(argv[0]) + " [OPTS] PROXY_MESH PROXY_CLOUD AIRSPACE_MESH OUT_VOLUME");
    args.set_description("TODO");
    args.add_option('r', "resolution", true, "guidance volume resolution [1.0]");
    args.add_option('\0', "max-distance", true, "maximum distance to surface [80.0]");
    args.add_option('\0', "max-altitude", true, "maximum altitude [100.0]");
    args.parse(argc, argv);

    Arguments conf;
    conf.proxy_mesh = args.get_nth_nonopt(0);
    conf.proxy_cloud = args.get_nth_nonopt(1);
    conf.airspace_mesh = args.get_nth_nonopt(2);
    conf.ovolume = args.get_nth_nonopt(3);
    conf.resolution = 1.0f;
    conf.max_distance = 80.0f;
    conf.max_altitude = 100.0f;

    for (util::ArgResult const* i = args.next_option();
         i != 0; i = args.next_option()) {
        switch (i->opt->sopt) {
        case 'r':
            conf.resolution = i->get_arg<float>();
        break;
        case '\0':
            if (i->opt->lopt == "max-distance") {
                conf.max_distance = i->get_arg<float>();
            } else if (i->opt->lopt == "max-altitude") {
                conf.max_altitude = i->get_arg<float>();
            } else {
                throw std::invalid_argument("Invalid option");
            }
        break;
        default:
            throw std::invalid_argument("Invalid option");
        }
    }

    return conf;
}

int main(int argc, char **argv) {
    util::system::register_segfault_handler();
    util::system::print_build_timestamp(argv[0]);

    Arguments args = parse_args(argc, argv);

    int device = cacc::select_cuda_device(3, 5);

    cacc::BVHTree<cacc::DEVICE>::Ptr dbvh_tree;
    {
        acc::BVHTree<uint, math::Vec3f>::Ptr bvh_tree;
        bvh_tree = load_mesh_as_bvh_tree(args.proxy_mesh);
        dbvh_tree = cacc::BVHTree<cacc::DEVICE>::create<uint, math::Vec3f>(bvh_tree);
    }
    cacc::tracing::bind_textures(dbvh_tree->cdata());

    mve::TriangleMesh::Ptr mesh;
    try {
        mesh = mve::geom::load_ply_mesh(args.airspace_mesh);
    } catch (std::exception& e) {
        std::cerr << "\tCould not load mesh: " << e.what() << std::endl;
        std::exit(EXIT_FAILURE);
    }

    std::vector<math::Vec3f> const & verts = mesh->get_vertices();

    //TODO merge with proxy mesh generation code
    acc::AABB<math::Vec3f> aabb = acc::calculate_aabb(verts);

    assert(acc::valid(aabb) && acc::volume(aabb) > 0.0f);

    int width = (aabb.max[0] - aabb.min[0]) / args.resolution + 1.0f;
    int height = (aabb.max[1] - aabb.min[1]) / args.resolution + 1.0f;
    int depth = args.max_altitude / args.resolution + 1.0f;

    std::cout << width << "x" << height << "x" << depth << std::endl;

    /* Create height map. */
    mve::FloatImage::Ptr hmap = mve::FloatImage::create(width, height, 1);
    hmap->fill(lowest);
    for (std::size_t i = 0; i < verts.size(); ++i) {
        math::Vec3f vertex = verts[i];
        int x = (vertex[0] - aabb.min[0]) / args.resolution + args.resolution / 2.0f;
        assert(0 <= x && x < width);
        int y = (vertex[1] - aabb.min[1]) / args.resolution + args.resolution / 2.0f;
        assert(0 <= y && y < height);
        float height = vertex[2];
        float z = hmap->at(x, y, 0);
        if (z > height) continue;

        hmap->at(x, y, 0) = height;
    }

    /* Estimate ground level and normalize height map */
    float ground_level = std::numeric_limits<float>::max();
    #pragma omp parallel for reduction(min:ground_level)
    for (int i = 0; i < hmap->get_value_amount(); ++i) {
        float height = hmap->at(i);
        if (height != lowest && height < ground_level) {
            ground_level = height;
        }
    }

    #pragma omp parallel for
    for (int i = 0; i < hmap->get_value_amount(); ++i) {
        float height = hmap->at(i);
        hmap->at(i) = (height != lowest) ? height - ground_level : 0.0f;
    }
    //ODOT merge with proxy mesh generation code

    mve::TriangleMesh::Ptr ocloud = mve::TriangleMesh::create();
    std::vector<math::Vec3f> & overts = ocloud->get_vertices();
    std::vector<uint> & ofaces = ocloud->get_faces();
    ofaces.push_back(width);
    ofaces.push_back(height);
    ofaces.push_back(depth);
    overts.resize(width * height * depth);
    std::vector<float> & ovalues = ocloud->get_vertex_values();
    ovalues.resize(width * height * depth, 0.0f);

    uint num_samples = 0;
    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x) {

            float px = (x - args.resolution / 2.0f) * args.resolution + aabb.min[0];
            float py = (y - args.resolution / 2.0f) * args.resolution + aabb.min[1];

            float fz = hmap->at(x, y, 0);

            for (int z = 0; z < depth; ++z) {
                float pz = ground_level + z * args.resolution;

                int idx = (z * height + y) * width + x;

                overts[idx] = math::Vec3f(px, py, pz);
                if (pz > fz) {
                    num_samples += 1;
                } else {
                    ovalues[idx] = -1.0f;
                }
            }
        }
    }

    uint num_verts;
    cacc::KDTree<3u, cacc::DEVICE>::Ptr dkd_tree;
    {
        mve::TriangleMesh::Ptr sphere = generate_sphere(1.0f, 3u);
        std::vector<math::Vec3f> const & verts = sphere->get_vertices();
        num_verts = verts.size();
        acc::KDTree<3u, uint>::Ptr kd_tree = acc::KDTree<3, uint>::create(verts);
        dkd_tree = cacc::KDTree<3u, cacc::DEVICE>::create<uint>(kd_tree);
    }
    cacc::nnsearch::bind_textures(dkd_tree->cdata());

    cacc::PointCloud<cacc::DEVICE>::Ptr dcloud;
    {
        cacc::PointCloud<cacc::HOST>::Ptr cloud;
        cloud = load_point_cloud(args.proxy_cloud);
        dcloud = cacc::PointCloud<cacc::DEVICE>::create<cacc::HOST>(cloud);
    }

    mve::CameraInfo cam;
    cam.flen = 0.86f;
    math::Matrix3f calib;

    std::string positions = litos(num_samples * 256ull * 90ull);
    std::string task = fmt::format("Sampling 5D volume at {} positions", positions);
    ProgressCounter counter(task, num_samples);

    #pragma omp parallel
    {
        cacc::set_cuda_device(device);

        hipStream_t stream;
        hipStreamCreate(&stream);

        int width = 1920;
        int height = 1080;
        cam.fill_calibration(calib.begin(), width, height);

        cacc::VectorArray<float, cacc::DEVICE>::Ptr dcon_hist;
        dcon_hist = cacc::VectorArray<float, cacc::DEVICE>::create(num_verts, 1, stream);
        dcon_hist->null();

        cacc::Image<float, cacc::DEVICE>::Ptr dhist;
        dhist = cacc::Image<float, cacc::DEVICE>::create(256, 90, stream);
        cacc::Image<float, cacc::HOST>::Ptr hist;
        hist = cacc::Image<float, cacc::HOST>::create(256, 90, stream);

        #pragma omp for schedule(dynamic)
        for (std::size_t i = 0; i < overts.size(); ++i) {
            if (ovalues[i] == -1.0f) continue;

            counter.progress<ETA>();

            dcon_hist->null();
            {
                dim3 grid(cacc::divup(dcloud->cdata().num_vertices, KERNEL_BLOCK_SIZE));
                dim3 block(KERNEL_BLOCK_SIZE);
                populate_histogram<<<grid, block, 0, stream>>>(
                    cacc::Vec3f(overts[i].begin()), args.max_distance,
                    dbvh_tree->cdata(), dcloud->cdata(), dkd_tree->cdata(),
                    dcon_hist->cdata());
            }

            {
                dim3 grid(cacc::divup(256, KERNEL_BLOCK_SIZE), 90);
                dim3 block(KERNEL_BLOCK_SIZE);
                evaluate_histogram<<<grid, block, 0, stream>>>(cacc::Mat3f(calib.begin()), width, height,
                    dkd_tree->cdata(), dcon_hist->cdata(), dhist->cdata());
            }

            *hist = *dhist;
            cacc::Image<float, cacc::HOST>::Data data = hist->cdata();

            hist->sync();

            float best = 0.0f;
            //#pragma omp parallel for reduction(max:best)
            for (int y = 0; y < data.height; ++y) {
                for (int x = 0; x < data.width; ++x) {
                    float v = data.data_ptr[y * data.pitch / sizeof(float) + x];
                    best = std::max(v, best);
                }
            }

            counter.inc();

            ovalues[i] = best;
        }
        hipStreamDestroy(stream);
    }


    mve::geom::SavePLYOptions opts;
    opts.write_vertex_values = true;
    mve::geom::save_ply_mesh(ocloud, args.ovolume, opts);

    return EXIT_SUCCESS;
}
