#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <unordered_set>

#include "util/system.h"
#include "util/arguments.h"
#include "util/file_system.h"

#include "mve/camera.h"
#include "mve/image_io.h"
#include "mve/mesh_io_ply.h"

#include "math/bspline.h"

#include "cacc/util.h"
#include "cacc/math.h"
#include "cacc/nnsearch.h"
#include "cacc/reduction.h"

#include "util/io.h"

#include "geom/sphere.h"
#include "geom/volume_io.h"

#include "utp/trajectory.h"
#include "utp/trajectory_io.h"

#include "eval/kernels.h"

#include "opti/nelder_mead.h"

struct Arguments {
    std::string in_trajectory;
    std::string proxy_mesh;
    std::string proxy_cloud;
    std::string out_trajectory;
    uint max_iters;
    float min_distance;
    float max_distance;
    float focal_length;
};

Arguments parse_args(int argc, char **argv) {
    util::Arguments args;
    args.set_exit_on_error(true);
    args.set_nonopt_minnum(4);
    args.set_nonopt_maxnum(4);
    args.set_usage("Usage: " + std::string(argv[0]) + " [OPTS] IN_TRAJECTORY PROXY_MESH PROXY_CLOUD OUT_TRAJECTORY");
    args.set_description("Optimize position and orientation of trajectory views.");
    args.add_option('\0', "min-distance", true, "minimum distance to surface [2.5]");
    args.add_option('\0', "max-distance", true, "maximum distance to surface [50.0]");
    args.add_option('\0', "focal-length", true, "camera focal length [0.86]");
    args.add_option('m', "max-iters", true, "maximum iterations [100]");
    args.parse(argc, argv);

    Arguments conf;
    conf.in_trajectory = args.get_nth_nonopt(0);
    conf.proxy_mesh = args.get_nth_nonopt(1);
    conf.proxy_cloud = args.get_nth_nonopt(2);
    conf.out_trajectory = args.get_nth_nonopt(3);
    conf.max_iters = 100;
    conf.max_distance = 50.0f;
    conf.min_distance = 2.5f;
    conf.focal_length = 0.86f;

    for (util::ArgResult const* i = args.next_option();
         i != 0; i = args.next_option()) {
        switch (i->opt->sopt) {
        case 'm':
            conf.max_iters = i->get_arg<uint>();
        break;
        case '\0':
            if (i->opt->lopt == "focal-length") {
                conf.focal_length = i->get_arg<float>();
            } else if (i->opt->lopt == "min-distance") {
                conf.min_distance = i->get_arg<float>();
            } else if (i->opt->lopt == "max-distance") {
                conf.max_distance = i->get_arg<float>();
            } else {
                throw std::invalid_argument("Invalid option");
            }
        break;
        default:
            throw std::invalid_argument("Invalid option");
        }
    }

    assert(conf.min_distance >= 0.0f);

    return conf;
}
float const pi = std::acos(-1.0f);

int main(int argc, char **argv) {
    util::system::register_segfault_handler();
    util::system::print_build_timestamp(argv[0]);

    Arguments args = parse_args(argc, argv);

    int device = cacc::select_cuda_device(3, 5);

    cacc::BVHTree<cacc::DEVICE>::Ptr dbvh_tree;
    {
        acc::BVHTree<uint, math::Vec3f>::Ptr bvh_tree;
        bvh_tree = load_mesh_as_bvh_tree(args.proxy_mesh);
        dbvh_tree = cacc::BVHTree<cacc::DEVICE>::create<uint, math::Vec3f>(bvh_tree);
    }

    uint num_sverts;
    cacc::KDTree<3u, cacc::DEVICE>::Ptr dkd_tree;
    {
        mve::TriangleMesh::Ptr sphere = generate_sphere(1.0f, 3u);
        std::vector<math::Vec3f> const & verts = sphere->get_vertices();
        num_sverts = verts.size();
        acc::KDTree<3u, uint>::Ptr kd_tree = acc::KDTree<3, uint>::create(verts);
        dkd_tree = cacc::KDTree<3u, cacc::DEVICE>::create<uint>(kd_tree);
    }
    cacc::nnsearch::bind_textures(dkd_tree->cdata());

    cacc::PointCloud<cacc::DEVICE>::Ptr dcloud;
    {
        cacc::PointCloud<cacc::HOST>::Ptr cloud;
        cloud = load_point_cloud(args.proxy_cloud);
        dcloud = cacc::PointCloud<cacc::DEVICE>::create<cacc::HOST>(cloud);
    }
    uint num_verts = dcloud->cdata().num_vertices;

    acc::KDTree<3, uint>::Ptr kd_tree(load_mesh_as_kd_tree(args.proxy_cloud));

    uint max_cameras = 50;

    cacc::VectorArray<cacc::Vec3f, cacc::DEVICE>::Ptr ddir_hist;
    ddir_hist = cacc::VectorArray<cacc::Vec3f, cacc::DEVICE>::create(num_verts, max_cameras);
    cacc::Array<float, cacc::DEVICE>::Ptr drecons;
    drecons = cacc::Array<float, cacc::DEVICE>::create(num_verts);

    mve::CameraInfo cam;
    cam.flen = args.focal_length;
    math::Matrix3f calib;
    int width = 1920;
    int height = 1080;
    cam.fill_calibration(calib.begin(), width, height);

    std::vector<mve::CameraInfo> trajectory;
    utp::load_trajectory(args.in_trajectory, &trajectory);
    std::vector<std::size_t> iters(trajectory.size(), args.max_iters);

    std::mt19937 gen(12345);

    std::vector<Simplex<3> > simplices(trajectory.size());

    std::vector<std::vector<float> > contribss(trajectory.size());

    float min_sq_distance = args.max_distance * args.max_distance;

    {
        std::normal_distribution<> pos_dist(0.0f, args.min_distance);
        for (std::size_t i = 0; i < trajectory.size(); ++i) {
            mve::CameraInfo const & cam = trajectory[i];
            math::Vec3f pos;
            cam.fill_camera_pos(pos.begin());

            std::array<math::Vector<float, 3>, 4> & verts = simplices[i].verts;
            for (std::size_t j = 0; j < verts.size(); ++j) {
                for (int k = 0; k < 3; ++k) {
                    verts[j][k] = pos[k] + pos_dist(gen);
                }
            }
        }
    }


    float avg_recon = 1.0f;

    std::vector<std::size_t> oindices;
    std::unordered_set<std::size_t> oidxset;
    #pragma omp parallel
    {
        cacc::set_cuda_device(device);

        hipStream_t stream;
        CHECK(hipStreamCreate(&stream));

        hipEvent_t event;
        CHECK(hipEventCreateWithFlags(&event, hipEventDefault | hipEventDisableTiming));

        cacc::VectorArray<float, cacc::DEVICE>::Ptr dcon_hist;
        dcon_hist = cacc::VectorArray<float, cacc::DEVICE>::create(num_sverts, 1, stream);

        cacc::Image<float, cacc::DEVICE>::Ptr dhist;
        dhist = cacc::Image<float, cacc::DEVICE>::create(128, 45, stream);
        cacc::Image<float, cacc::HOST>::Ptr hist;
        hist = cacc::Image<float, cacc::HOST>::create(128, 45, stream);

        for (uint i = 0; i < args.max_iters; ++i) {
            #pragma omp single
            {
                oidxset.clear();
                oindices.clear();
                //drecons->null();
                ddir_hist->clear();

                std::discrete_distribution<> d(iters.begin(), iters.end());
                {
                    std::vector<math::Vec3f> poss;
                    for (std::size_t j = 0; j < trajectory.size(); ++j) {
                        std::size_t idx = d(gen);

                        math::Vec3f pos;
                        trajectory[idx].fill_camera_pos(pos.begin());

                        bool too_close = std::any_of(poss.begin(), poss.end(),
                            [&pos, &min_sq_distance](math::Vec3f const & opos) -> bool {
                                return (pos - opos).square_norm() < min_sq_distance;
                        });

                        if (!too_close) {
                            oindices.push_back(idx);
                            poss.push_back(pos);
                            iters[idx] -= 1;
                        }
                    }
                }
                oidxset.insert(oindices.begin(), oindices.end());
            }

            #pragma omp for schedule(dynamic)
            for (std::size_t j = 0; j < trajectory.size(); ++j) {
                if (oidxset.count(j)) continue;
                mve::CameraInfo const & cam = trajectory[j];

                math::Vec3f pos;
                cam.fill_camera_pos(pos.begin());
                math::Matrix4f w2c;
                cam.fill_world_to_cam(w2c.begin());

                dim3 grid(cacc::divup(num_verts, KERNEL_BLOCK_SIZE));
                dim3 block(KERNEL_BLOCK_SIZE);
                populate_direction_histogram<<<grid, block, 0, stream>>>(
                    cacc::Vec3f(pos.begin()), args.max_distance,
                    cacc::Mat4f(w2c.begin()), cacc::Mat3f(calib.begin()),
                    width, height,
                    dbvh_tree->accessor(),
                    dcloud->cdata(), ddir_hist->cdata()
                );

                cacc::sync(stream, event, std::chrono::microseconds(100));
            }
            ((void)0);

            #pragma omp single
            {
                {
                    dim3 grid(cacc::divup(num_verts, 2));
                    dim3 block(32, 2);
                    sort_direction_histogram<<<grid, block, 0, stream>>>(
                        ddir_hist->cdata());
                }

                {
                    dim3 grid(cacc::divup(num_verts, KERNEL_BLOCK_SIZE));
                    dim3 block(KERNEL_BLOCK_SIZE);
                    evaluate_direction_histogram<<<grid, block, 0, stream>>>(
                        ddir_hist->cdata(), drecons->cdata());
                }

                cacc::sync(stream, event, std::chrono::microseconds(100));
            }

            #pragma omp for schedule(dynamic)
            for (std::size_t j = 0; j < oindices.size(); ++j) {
                std::size_t idx = oindices[j];
                mve::CameraInfo & cam = trajectory[idx];

                float vmax = 0.0f;
                float vtheta = 0.0f;
                float vphi = 0.0f;

                std::function<float(math::Vec3f)> func =
                    [&] (math::Vec3f const & pos) -> float
                {
                    if (pos[2] < args.min_distance) return 0.0f;
                    if (kd_tree->find_nn(pos, nullptr, args.min_distance)) return 0.0f;

                    dcon_hist->null();
                    {
                        dim3 grid(cacc::divup(num_verts, KERNEL_BLOCK_SIZE));
                        dim3 block(KERNEL_BLOCK_SIZE);
                        populate_histogram<<<grid, block, 0, stream>>>(
                            cacc::Vec3f(pos.begin()), args.max_distance, avg_recon,
                            dbvh_tree->accessor(), dcloud->cdata(), dkd_tree->cdata(),
                            ddir_hist->cdata(), drecons->cdata(), dcon_hist->cdata());
                    }

                    {
                        dim3 grid(cacc::divup(128, KERNEL_BLOCK_SIZE), 45);
                        dim3 block(KERNEL_BLOCK_SIZE);
                        evaluate_histogram<<<grid, block, 0, stream>>>(
                            cacc::Mat3f(calib.begin()), width, height,
                            dkd_tree->cdata(), dcon_hist->cdata(), dhist->cdata());
                    }

                    *hist = *dhist;
                    cacc::Image<float, cacc::HOST>::Data data = hist->cdata();

                    cacc::sync(stream, event, std::chrono::microseconds(100));

                    float max = 0.0f;
                    float theta = 0.0f;
                    float phi = 0.0f;

                    for (int y = 0; y < data.height; ++y) {
                        for (int x = 0; x < data.width; ++x) {
                            float v = data.data_ptr[y * data.pitch / sizeof(float) + x];
                            if (v > max) {
                                max = v;
                                theta = (0.5f + (y / (float) data.height) / 2.0f) * pi;
                                phi = (x / (float) data.width) * 2.0f * pi;
                            }
                        }
                    }

                    if (max > vmax) {
                        vmax = max;
                        vtheta = theta;
                        vphi = phi;
                    }

                    return -max;
                };

                Simplex<3> & simplex  = simplices[idx];

                float value;
                std::size_t vid;
                std::tie(vid, value) = nelder_mead(&simplex, func);

                math::Vec3f pos = simplex.verts[vid];

                math::Matrix3f rot = utp::rotation_from_spherical(vtheta, vphi);
                math::Vec3f trans = -rot * pos;

                std::copy(trans.begin(), trans.end(), cam.trans);
                std::copy(rot.begin(), rot.end(), cam.rot);
            }

            #pragma omp for schedule(dynamic)
            for (std::size_t j = 0; j < oindices.size(); ++j) {
                mve::CameraInfo const & cam = trajectory[oindices[j]];

                math::Vec3f pos;
                cam.fill_camera_pos(pos.begin());
                math::Matrix4f w2c;
                cam.fill_world_to_cam(w2c.begin());

                {
                    dim3 grid(cacc::divup(num_verts, KERNEL_BLOCK_SIZE));
                    dim3 block(KERNEL_BLOCK_SIZE);
                    populate_direction_histogram<<<grid, block, 0, stream>>>(
                        cacc::Vec3f(pos.begin()), args.max_distance,
                        cacc::Mat4f(w2c.begin()), cacc::Mat3f(calib.begin()),
                        width, height,
                        dbvh_tree->accessor(), dcloud->cdata(),
                        ddir_hist->cdata()
                    );
                }

                cacc::sync(stream, event, std::chrono::microseconds(100));
            }

            #pragma omp single
            {
                {
                    dim3 grid(cacc::divup(num_verts, 2));
                    dim3 block(32, 2);
                    sort_direction_histogram<<<grid, block, 0, stream>>>(
                        ddir_hist->cdata());
                }

                {
                    dim3 grid(cacc::divup(num_verts, KERNEL_BLOCK_SIZE));
                    dim3 block(KERNEL_BLOCK_SIZE);
                    evaluate_direction_histogram<<<grid, block, 0, stream>>>(
                        ddir_hist->cdata(), drecons->cdata());
                }

                cacc::sync(stream, event, std::chrono::microseconds(100));

                //float length = utp::length(trajectory);

                avg_recon = cacc::sum(drecons) / num_verts;
                //std::cout << i << "(" << oindices.size() << ") " << avg_recon << " " << length << std::endl;
                std::cout << i << "(" << oindices.size() << ") " << avg_recon << std::endl;
            }
        }
        hipEventDestroy(event);
        hipStreamDestroy(stream);
    }

    utp::save_trajectory(trajectory, args.out_trajectory);

    return EXIT_SUCCESS;
}
